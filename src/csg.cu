#include "hip/hip_runtime.h"
#include <cmath>
using namespace std;
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <opencv2/opencv.hpp>
using namespace cv;
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>

#define TWO_PI 6.28318530718f
#define W 512
#define H 512
#define N 64
#define MAX_STEP 64
#define MAX_DISTANCE 2.0f
#define EPSILON 1e-6f

#define block_x 32

struct Result {
	float sd, emissive;
};

__device__
Result unionOp(Result a, Result b) {
	return a.sd < b.sd ? a : b;
}

__device__
Result intersectOp(Result a, Result b) {
	Result r = a.sd > b.sd ? b : a;
	r.sd = a.sd > b.sd ? a.sd : b.sd;
	return r;
}

__device__
Result subtractOp(Result a, Result b) {
	Result r = a;
	r.sd = (a.sd > -b.sd) ? a.sd : -b.sd;
	return r;
}

__device__
Result complementOp(Result a) {
	a.sd = -a.sd;
	return a;
}

__device__
float CircleSDF(float x, float y, float cx, float cy, float r) {
	float ux = x - cx, uy = y - cy;
	return sqrtf(ux * ux + uy * uy) - r;
}

__device__
Result Scene(float x, float y) {
#if 0
    Result r1 = { CircleSDF(x, y, 0.3f, 0.3f, 0.10f), 2.0f };
    Result r2 = { CircleSDF(x, y, 0.3f, 0.7f, 0.05f), 0.8f };
    Result r3 = { CircleSDF(x, y, 0.7f, 0.5f, 0.10f), 0.0f };
    return unionOp(unionOp(r1, r2), r3);
#else
    Result a = { CircleSDF(x, y, 0.4f, 0.5f, 0.20f), 1.0f };
    Result b = { CircleSDF(x, y, 0.6f, 0.5f, 0.20f), 0.8f };
    return unionOp(a, b);
    // return intersectOp(a, b);
    // return subtractOp(a, b);
    // return subtractOp(b, a);
#endif
}

__device__
float Trace(float ox, float oy, float dx, float dy) {
	float t = 0.001f;
	for (int i = 0; i < MAX_STEP && t < MAX_DISTANCE; i++) {
		Result r = Scene(ox + dx * t, oy + dy * t);
		if (r.sd < EPSILON)
			return r.emissive;
		t += r.sd;
	}
	return 0.f;
}

__global__
void Sample(hiprandState *rand_states, float *buffer) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= W || y >= H) return;
	int offset = x + y * W;

	float sum = 0.0f;
	for (int i = 0; i < N; i++) {
		float a = TWO_PI * (i + hiprand_uniform(rand_states + offset)) / N;
		sum += Trace(float(x) / W, float(y) / H, cos(a), sin(a));
	}
	buffer[offset * 3 + 0] = sum / N * 255;
	buffer[offset * 3 + 1] = sum / N * 255;
	buffer[offset * 3 + 2] = sum / N * 255;
}

__global__
void InitRandStates(hiprandState *rand_states, long seed) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= W || y >= H) return;
	int offset = x + y * blockDim.x * gridDim.x;

	hiprand_init(seed, offset, 0, rand_states + offset);
}

int main() {
	float *buffer;
	hipMalloc(&buffer, W * H * 3 * sizeof(float));

	hiprandState *rand_states;
	hipMalloc(&rand_states, W * H * sizeof(hiprandState));
	InitRandStates<<<dim3((W-1)/block_x+1, (H-1)/block_x+1), dim3(block_x, block_x)>>>(rand_states, time(NULL));

	Sample<<<dim3((W-1)/block_x+1, (H-1)/block_x+1), dim3(block_x, block_x)>>>(rand_states, buffer);

	Mat img = Mat(H, W, CV_32FC3);
	hipMemcpy(img.data, buffer, W * H * 3 * sizeof(float), hipMemcpyDeviceToHost);
	imwrite("csg.png", img);
}
