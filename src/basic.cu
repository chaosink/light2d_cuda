#include "hip/hip_runtime.h"
#include <cmath>
using namespace std;
#include "GL/glew.h"
#include "GLFW/glfw3.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>

#define TWO_PI 6.28318530718f
#define W 512
#define H 512
#define N 2048
#define MAX_STEP 8
#define MAX_DISTANCE 2.0f
#define EPSILON 1e-6f

#define block_x 32

__device__
float circleSDF(float x, float y, float cx, float cy, float r) {
	float ux = x - cx, uy = y - cy;
	return sqrtf(ux * ux + uy * uy) - r;
}

__device__
float Trace(float ox, float oy, float dx, float dy) {
	float t = 0.0f;
	for (int i = 0; i < MAX_STEP && t < MAX_DISTANCE; i++) {
		float sd = circleSDF(ox + dx * t, oy + dy * t, 0.5f, 0.5f, 0.1f);
		if (sd < EPSILON)
			return 2.0f;
		t += sd;
	}
	return 0.0f;
}

__global__
void InitRandStates(hiprandState *rand_states, long seed) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= W || y >= H) return;
	int offset = x + y * blockDim.x * gridDim.x;

	hiprand_init(seed, offset, 0, rand_states + offset);
}

__global__
void Sample(hiprandState *rand_states, float *buffer) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= W || y >= H) return;
	int offset = x + y * W;

	float sum = 0.0f;
	for (int i = 0; i < N; i++) {
		float a = TWO_PI * (i + hiprand_uniform(rand_states + offset)) / N;
		sum += Trace(float(x) / W, float(y) / H, cosf(a), sinf(a));
	}
	buffer[offset * 3 + 0] = sum / N * 255;
	buffer[offset * 3 + 1] = sum / N * 255;
	buffer[offset * 3 + 2] = sum / N * 255;
}

int main() {
	float *buffer;
	hipMalloc(&buffer, W * H * 3 * sizeof(float));

	hiprandState *rand_states;
	hipMalloc(&rand_states, W * H * sizeof(hiprandState));
	InitRandStates<<<dim3((W-1)/block_x+1, (H-1)/block_x+1), dim3(block_x, block_x)>>>(rand_states, time(NULL));

	Sample<<<dim3((W-1)/block_x+1, (H-1)/block_x+1), dim3(block_x, block_x)>>>(rand_states, buffer);

	float image[W * H * 3];
	hipMemcpy(image, buffer, sizeof(image), hipMemcpyDeviceToHost);

	uint8_t output[W * H * 3];
	for(int i = 0; i < H; ++i)
		for(int j = 0; j < W; ++j)
			for(int k = 0; k < 3; ++k) {
				output[(i * W + j) * 3 + k] = image[(i * W + j) * 3 + k];
			}
	stbi_write_png("baisc.png", W, H, 3, output, 0);
}
